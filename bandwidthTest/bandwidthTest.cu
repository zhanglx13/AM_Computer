#include "hip/hip_runtime.h"
/*
 * This is a simple test program to measure the memcopy bandwidth of the GPU.
 * It can measure device to device copy bandwidth, host to device copy bandwidth
 * for pageable and pinned memory, and device to host copy bandwidth for pageable
 * and pinned memory.
 *
 * Usage:
 * ./bandwidthTest [option]...
 */
/* A little information about write combined memory
 *  By default page-locked host memory is allocated as cacheable. 
 *  It can optionally be allocated as write-combining instead by 
 *  passing flag hipHostMallocWriteCombined to hipHostAlloc(). 
 *  Write-combining memory frees up the host's L1 and L2 cache 
 *  resources, making more cache available to the rest of the 
 *  application. In addition, write-combining memory is not 
 *  snooped during transfers across the PCI Express bus, which 
 *  can improve transfer performance by up to 40%.
 *  Reading from write-combining memory from the host is prohibitively 
 *  slow, so write-combining memory should in general be used for 
 *  memory that the host only writes to. 
 */

// CUDA runtime
#include <hip/hip_runtime.h>

//includes
#include <helper_functions.h>    // helper for shared functions common to CUDA Samples
#include <hip/hip_runtime_api.h>        // helper functions for CUDA error checking and initialization

#include <hip/hip_runtime.h>

#include <memory>
#include <iostream>
#include <cassert>

// defines, project
#define MEMORY_ITERATIONS   10
#define DEFAULT_SIZE        ( 32 * ( 1 << 20) )     //32 M
#define DEFAULT_INCREMENT   ( 1 << 22 )             //4 M
#define CACHE_CLEAR_SIZE    ( 1 << 24 )             //16 M

// defines, experiment
#define MEMSIZE_MAX         ( 1 << 26 )             //64 M
#define MEMSIZE_START       ( 1 << 10 )             //1 KB
#define INCREMENT_1KB       ( 1 << 10 )             //1 KB
#define INCREMENT_2KB       ( 1 << 11 )             //2 KB
#define INCREMENT_4KB       ( 1 << 12 )             //4 KB
#define INCREMENT_8KB       ( 1 << 13 )             //8 KB

// enums, project
enum memcpyKind { DEVICE_TO_HOST, HOST_TO_DEVICE, DEVICE_TO_DEVICE };
enum printMode  { USER_READABLE, CSV };
enum memoryMode { PINNED, PAGEABLE };

const char *sMemoryCopyKind[] =
{
    "Device to Host",
    "Host to Device",
    "Device to Device",
    NULL
};

const char *sMemoryMode[] =
{
    "PINNED",
    "PAGEABLE",
    NULL
};


// if true, use CPU based timing for everything
static bool bDontUseGPUTiming;

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
float testHostToDeviceTransfer(unsigned int memSize, memoryMode memMode, bool wc);
float testDeviceToDeviceTransfer(unsigned int);
void printResultsReadable(unsigned int *memSizes, double *bandwidths, unsigned int count, memcpyKind kind, memoryMode memMode, bool wc);
void printResultsCSV(unsigned int *memSizes, double *bandwidths, unsigned int count, memcpyKind kind, memoryMode memMode, bool wc);
void printHelp(void);
////////////////////////////////////////////////////////////////////////////////


int main(int argc, char **argv)
{
    printf("D2D: %lf\n",testDeviceToDeviceTransfer(DEFAULT_SIZE));
    printf("H2D: %lf\n",testHostToDeviceTransfer(DEFAULT_SIZE, PINNED, true));
    printf("H2D: %lf\n",testHostToDeviceTransfer(DEFAULT_SIZE, PAGEABLE, false));
    return 0;
}


///////////////////////////////////////////////////////////////////////////////
//! test the bandwidth of a host to device memcopy of a specific size
///////////////////////////////////////////////////////////////////////////////
float testHostToDeviceTransfer(unsigned int memSize, memoryMode memMode, bool wc)
{
    StopWatchInterface *timer = NULL;
    float elapsedTimeInMs = 0.0f;
    float bandwidthInMBs = 0.0f;
    hipEvent_t start, stop;

    sdkCreateTimer(&timer);
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    // allocate host memory
    unsigned char *h_idata = NULL;

    if (memMode == PINNED)
        checkCudaErrors(hipHostAlloc((void**)&h_idata, memSize, (wc) ? hipHostMallocWriteCombined : 0));
    else{
        h_idata = (unsigned char*)malloc(memSize);
        if (h_idata == 0){
            fprintf(stderr, "Not enough memory on host to run test!\n");
            exit(EXIT_FAILURE);
        }
    }

    // initialize host memory
    for (unsigned int i = 0 ; i < memSize/sizeof(unsigned char) ; i ++)
        h_idata[i] = (unsigned char)(i & 0xff);

    // allocate device memory
    unsigned char *d_odata;
    checkCudaErrors(hipMalloc((void**)&d_odata, memSize));

    sdkStartTimer(&timer);
    checkCudaErrors(hipEventRecord(start, 0));

    // copy data from host to device
    if (PINNED == memMode)
        for (unsigned int i = 0 ; i < MEMORY_ITERATIONS ; i ++)
            checkCudaErrors(hipMemcpyAsync(d_odata, h_idata, memSize,
                                            hipMemcpyHostToDevice, 0));
    else 
        for (unsigned int i = 0 ; i < MEMORY_ITERATIONS ; i ++)
            checkCudaErrors(hipMemcpy(d_odata, h_idata, memSize,
                                       hipMemcpyHostToDevice));

    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipDeviceSynchronize());
    sdkStopTimer(&timer);
    checkCudaErrors(hipEventElapsedTime(&elapsedTimeInMs, start, stop));
    
    if (PAGEABLE == memMode || bDontUseGPUTiming)
        elapsedTimeInMs = sdkGetTimerValue(&timer);

    sdkResetTimer(&timer);
    
    // calculate the bandwidth
    bandwidthInMBs = ((float)(1<<10) * memSize * MEMORY_ITERATIONS) / ((float)(1<<20) * elapsedTimeInMs);

    // clean up memory
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
    sdkDeleteTimer(&timer);

    if (PINNED == memMode)
        checkCudaErrors(hipHostFree(h_idata));
    else 
        free(h_idata);

    checkCudaErrors(hipFree(d_odata));

    return bandwidthInMBs;







}


///////////////////////////////////////////////////////////////////////////////
//! test the bandwidth of a device to device memcopy of a specific size
///////////////////////////////////////////////////////////////////////////////
float testDeviceToDeviceTransfer(unsigned int memSize)
{
    StopWatchInterface *timer = NULL;
    float elapsedTimeInMs = 0.0f;
    float bandwidthInMBs = 0.0f;
    hipEvent_t start, stop;

    sdkCreateTimer(&timer);
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    // allocate host memory
    unsigned char *h_idata = (unsigned char*)malloc(memSize);
    if( h_idata == 0){
        fprintf(stderr, "Not enough memory available on host to run test!\n");
        exit(EXIT_FAILURE);
    }

    // initialize the host memory
    for (unsigned int i = 0 ; i < memSize/sizeof(unsigned char) ; i ++)
        h_idata[i] = (unsigned char)(i & 0xff);

    // allocate device memory
    unsigned char *d_idata, *d_odata;
    checkCudaErrors(hipMalloc((void**)&d_idata, memSize));
    checkCudaErrors(hipMalloc((void**)&d_odata, memSize));

    // initialize input device memory
    checkCudaErrors(hipMemcpy(d_idata, h_idata, memSize,
                               hipMemcpyHostToDevice));

    // run the memcpy
    sdkStartTimer(&timer);
    checkCudaErrors(hipEventRecord(start, 0));

    for (unsigned int i = 0 ; i < MEMORY_ITERATIONS ; i ++)
        checkCudaErrors(hipMemcpy(d_odata, d_idata, memSize,
                                   hipMemcpyDeviceToDevice));

    checkCudaErrors(hipEventRecord(stop, 0));
    
    // since device to device memory copies are non-blocking,
    // hipDeviceSynchronize() is required in order to get
    // proper timing
    checkCudaErrors(hipDeviceSynchronize());

    // get the total elapsed time in ms
    sdkStopTimer(&timer);
    checkCudaErrors(hipEventElapsedTime(&elapsedTimeInMs, start, stop));

    if(bDontUseGPUTiming)
        elapsedTimeInMs = sdkGetTimerValue(&timer);

    // calculate the bandwidth in MB/s
    bandwidthInMBs = 2.0f * ((float)(1<<10) * memSize * MEMORY_ITERATIONS) /
                    (elapsedTimeInMs * (float)(1<<20));

    // clean up memory
    sdkDeleteTimer(&timer);
    free(h_idata);
    checkCudaErrors(hipFree(d_idata));
    checkCudaErrors(hipFree(d_odata));
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

    return bandwidthInMBs;

}

/////////////////////////////////////////////////////////
//print results in an easily read format
////////////////////////////////////////////////////////
void printResultsReadable(unsigned int *memSizes, double *bandwidths, unsigned int count, memcpyKind kind, memoryMode memMode, bool wc)
{
    printf(" %s Bandwidth\n", sMemoryCopyKind[kind]);
    printf(" %s Memory Transfers\n", sMemoryMode[memMode]);

    if (wc)
        printf(" Write-Combined Memory Writes are Enabled");

    printf("   Transfer Size (Bytes)\tBandwidth(MB/s)\n");
    unsigned int i;

    for (i = 0; i < (count - 1); i++)
    {
        printf("   %u\t\t\t%s%.1f\n", memSizes[i], (memSizes[i] < 10000)? "\t" : "", bandwidths[i]);
    }

    printf("   %u\t\t\t%s%.1f\n\n", memSizes[i], (memSizes[i] < 10000)? "\t" : "", bandwidths[i]);
}


///////////////////////////////////////////////////////////////////////////
//print results in a database format
///////////////////////////////////////////////////////////////////////////
void printResultsCSV(unsigned int *memSizes, double *bandwidths, unsigned int count, memcpyKind kind, memoryMode memMode, bool wc)
{
    std::string sConfig;

    // log config information
    if (kind == DEVICE_TO_DEVICE)
        sConfig += "D2D";
    else
    {
        if (kind == DEVICE_TO_HOST)
            sConfig += "D2H";
        else if (kind == HOST_TO_DEVICE)
            sConfig += "H2D";

        if (memMode == PAGEABLE)
            sConfig += "-Paged";
        else if (memMode == PINNED)
        {
            sConfig += "-Pinned";

            if (wc)
                sConfig += "WriteCombined";
        }
    }
    unsigned int i;
    double dSeconds = 0.0;

    for (i = 0 ; i < count ; i ++){
        dSeconds = (double)memSizes[i] / (bandwidths[i] * (double)(1<<20));
        printf("bandwidthTest-%s, Bandwidth = %.1f MB/s, Time = %.5f s, Size = %u bytes\n",
                sConfig.c_str(), bandwidths[i], dSeconds, memSizes[i]);
    }
}

///////////////////////////////////////////////////////////////////////////
//Print help screen
///////////////////////////////////////////////////////////////////////////
void printHelp(void)
{
    printf("Usage:  bandwidthTest [OPTION]...\n");
    printf("Test the bandwidth for device to host, host to device, and device to device transfers\n");
    printf("\n");
    printf("Example:  measure the bandwidth of device to host pinned memory copies in the range 1024 Bytes to 102400 Bytes in 1024 Byte increments     \n");
    printf("./bandwidthTest --memory=pinned --start=1024 --end=102400 --increment=1024 --dtoh\n");

    printf("\n");
    printf("Options:\n");
    printf("--help\tDisplay this help menu\n");
    printf("--csv\tPrint results as a CSV\n");
    /* We use device 0 */
//    printf("--device=[deviceno]\tSpecify the device device to be used\n");
//    printf("  all - compute cumulative bandwidth on all the devices\n");
//    printf("  0,1,2,...,n - Specify any particular device to be used\n");
    printf("--memory=[MEMMODE]\tSpecify which memory mode to use\n");
    printf("  pageable - pageable memory\n");
    printf("  pinned   - non-pageable system memory\n");
    /* We use range*/
//    printf("--mode=[MODE]\tSpecify the mode to use\n");
//    printf("  quick - performs a quick measurement\n");
//    printf("  range - measures a user-specified range of values\n");
//    printf("  shmoo - performs an intense shmoo of a large range of values\n");
    printf("--htod\tMeasure host to device transfers\n");
    printf("--dtoh\tMeasure device to host transfers\n");
    printf("--dtod\tMeasure device to device transfers\n");
#if CUDART_VERSION >= 2020
    printf("--wc\tAllocate pinned memory as write-combined\n");
#endif
    printf("--cputiming\tForce CPU-based timing always\n");

    printf("Range mode options\n");
    printf("--start=[SIZE]\tStarting transfer size in bytes\n");
    printf("--end=[SIZE]\tEnding transfer size in bytes\n");
    printf("--increment=[SIZE]\tIncrement size in bytes\n");
}
